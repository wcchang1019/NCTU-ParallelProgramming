/**********************************************************************
 * DESCRIPTION:
 *   Serial Concurrent Wave Equation - C Version
 *   This program implements the concurrent wave equation
 *********************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265
#define MAXTHREADSIZE 500

void check_param(void);
void init_line(void);
void update (void);
void printfinal (void);

int nsteps,                   /* number of time steps */
    tpoints,            /* total points along string */
    rcode;                    /* generic return code */
float  values[MAXPOINTS+2];   /* values at time t */

float *cuda_values;
/**********************************************************************
 * Checks input values from parameters
 *********************************************************************/
void check_param(void)
{
   char tchar[20];

   /* check number of points, number of iterations */
   while ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS)) {
      printf("Enter number of points along vibrating string [%d-%d]: "
           ,MINPOINTS, MAXPOINTS);
      scanf("%s", tchar);
      tpoints = atoi(tchar);
      if ((tpoints < MINPOINTS) || (tpoints > MAXPOINTS))
         printf("Invalid. Please enter value between %d and %d\n", 
                 MINPOINTS, MAXPOINTS);
   }
   while ((nsteps < 1) || (nsteps > MAXSTEPS)) {
      printf("Enter number of time steps [1-%d]: ", MAXSTEPS);
      scanf("%s", tchar);
      nsteps = atoi(tchar);
      if ((nsteps < 1) || (nsteps > MAXSTEPS))
         printf("Invalid. Please enter value between 1 and %d\n", MAXSTEPS);
   }

   printf("Using points = %d, steps = %d\n", tpoints, nsteps);

}

/**********************************************************************
 *     Initialize points on line
 *********************************************************************/
void init_line(void)
{
   int j;
   float x, fac, k, tmp;

   /* Calculate initial values based on sine curve */
   fac = 2.0 * PI;
   k = 0.0; 
   tmp = tpoints - 1;
   for (j = 1; j <= tpoints; j++) {
      x = k/tmp;
      values[j] = sin (fac * x);
      k = k + 1.0;
   } 
}

/**********************************************************************
 *      Calculate new values using wave equation
 *********************************************************************/
__device__ float do_math(float now_value, float old_value)
{
   float dtime, c, dx, tau, sqtau;

   dtime = 0.3;
   c = 1.0;
   dx = 1.0;
   tau = (c * dtime / dx);
   sqtau = tau * tau;
   return (2.0 * now_value) - old_value + (sqtau *  (-2.0)*now_value);
}

/**********************************************************************
 *     Update all values along line a specified number of times
 *********************************************************************/
__global__ void update(float *cuda_values, int nsteps, int tpoints)
{
   int i, j;
   j = blockIdx.x*MAXTHREADSIZE + threadIdx.x + 1;

   /* Update values for each time step */
   if(j <= tpoints){
      float now_value = cuda_values[j];
      float old_value = now_value;
      float new_value;
      for (i = 1; i<= nsteps; i++) {
         if ((j == 1) || (j  == tpoints))
            new_value = 0.0;
         else{
            new_value = do_math(now_value, old_value);
         }
         old_value = now_value;
         now_value = new_value;
      }
      cuda_values[j] = now_value;
   }
}

/**********************************************************************
 *     Print final results
 *********************************************************************/
void printfinal()
{
   int i;

   for (i = 1; i <= tpoints; i++) {
      printf("%6.4f ", values[i]);
      if (i%10 == 0)
         printf("\n");
   }
}

/**********************************************************************
 * Main program
 *********************************************************************/
int main(int argc, char *argv[])
{
   sscanf(argv[1],"%d",&tpoints);
   sscanf(argv[2],"%d",&nsteps);
   check_param();
   printf("Initializing points on the line...\n");
   init_line();
   hipMalloc(&cuda_values, sizeof(float)*(MAXPOINTS+2));
   hipMemcpy(cuda_values, values, sizeof(float)*(MAXPOINTS+2), hipMemcpyHostToDevice);
   printf("Updating all points for all time steps...\n");
   int block_size = ceil((float)tpoints/MAXTHREADSIZE);
   update<<<block_size, MAXTHREADSIZE>>>(cuda_values, nsteps, tpoints);
   hipMemcpy(values, cuda_values, sizeof(float)*(MAXPOINTS+2), hipMemcpyDeviceToHost);
   printf("Printing final results...\n");
   printfinal();
   printf("\nDone.\n\n");
   hipFree(cuda_values);
   return 0;
}